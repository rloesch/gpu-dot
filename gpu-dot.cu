#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>


int main(int argc, char const *argv[])
{
    /* Number of elements */
    const int nvals = 1024;

    /* Number of bytes to store nvals elements */
    const size_t sz = sizeof(double) * (size_t)nvals;

    /* Host vectors */
    double x[nvals], y[nvals];

    /* Device vectors */
    double *x_, *y_;

    /* Host result */
    double result = 0.0;


    /* Initialize vectors */
    for(int i=0; i < nvals; i++) {
        x[i] = 1.0;
        y[i] = 1.0;
    }

    printf("Initializing two vectors with %d elements (%d bytes) each\n", nvals, sz);

    /* Create the CUBLAS library context */
    hipblasHandle_t h;
    int cublasCreateRV = hipblasCreate(&h);
	
	if (cublasCreateRV == HIPBLAS_STATUS_NOT_INITIALIZED)
		printf("CUBLAS_STATUS_NOT_INITIALIZED\n");
	else if (cublasCreateRV == HIPBLAS_STATUS_SUCCESS)
		printf("CUBLAS_STATUS_SUCCESS\n");
	else if (cublasCreateRV == HIPBLAS_STATUS_ALLOC_FAILED)
		printf("CUBLAS_STATUS_ALLOC_FAILED\n");
	else
		printf("Invalid hipblasCreate return value.\n");

    /* Allocate memory for both vectors on the device */
    hipMalloc( (void **)(&x_), sz);
    hipMalloc( (void **)(&y_), sz);

    /* Copy the vectors from the host to the device */
    hipMemcpy(x_, x, sz, hipMemcpyHostToDevice);
    hipMemcpy(y_, y, sz, hipMemcpyHostToDevice);

    /* Run the dot calculation on the device using CUBLAS */
    int cublasReturn = hipblasDdot(h, nvals, x_, 1, y_, 1, &result);

    /* Print result */
	if (cublasReturn == HIPBLAS_STATUS_SUCCESS)
		printf("CUBLAS_STATUS_SUCCESS\n");
	else if (cublasReturn == HIPBLAS_STATUS_NOT_INITIALIZED)
		printf("CUBLAS_STATUS_NOT_INITIALIZED\n");
	else if (cublasReturn == HIPBLAS_STATUS_ALLOC_FAILED)
		printf("CUBLAS_STATUS_ALLOC_FAILED\n");
	else if (cublasReturn == HIPBLAS_STATUS_ARCH_MISMATCH)
		printf("CUBLAS_STATUS_ARCH_MISMATCH\n");
	else if (cublasReturn == HIPBLAS_STATUS_EXECUTION_FAILED)
		printf("CUBLAS_STATUS_EXECUTION_FAILED\n");
	else
		printf("Invalid cublas response.\n");
	
    printf("%.3f\n", result);

    return 0;
}
