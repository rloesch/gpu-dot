#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>


int main(int argc, char const *argv[])
{
    /* Number of elements */
    const int nvals = 1024;

    /* Number of bytes to store nvals elements */
    const size_t sz = sizeof(double) * (size_t)nvals;

    /* Host vectors */
    double x[nvals], y[nvals];

    /* Device vectors */
    double *x_, *y_;

    /* Host result */
    double result = 0.0;


    /* Initialize vectors */
    for(int i=0; i < nvals; i++) {
        x[i] = 1.0;
        y[i] = 1.0;
    }

    printf("Initializing two vectors with %d elements (%d bytes) each\n", nvals, sz);

    /* Create the CUBLAS library context */
    hipblasHandle_t h;
    hipblasCreate(&h);

    /* Allocate memory for both vectors on the device */
    hipMalloc( (void **)(&x_), sz);
    hipMalloc( (void **)(&y_), sz);

    /* Copy the vectors from the host to the device */
    hipMemcpy(x_, x, sz, hipMemcpyHostToDevice);
    hipMemcpy(y_, y, sz, hipMemcpyHostToDevice);

    /* Run the dot calculation on the device using CUBLAS */
    hipblasDdot(h, nvals, x_, 1, y_, 1, &result);

    /* Print result */
    printf("%.3f\n", result);

    return 0;
}
